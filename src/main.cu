#include <GL/glew.h>
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <GL/glui.h>

#include "util/input.cuh"
#include "sim.cuh"

static SimState *global_state = NULL;
static SimState state;

#define NUM_MATERIALS 10

void display_wrapper()
{
    if (global_state)
    {
        display(global_state);
    }
}

int main(int argc, char **argv)
{
    
    material material_list[NUM_MATERIALS] = {
        {8.854e-12, 1.2566e-6, 0.0, 0.0f, 0.0f, 0.0f, "Vacuum"},        // VACUUM
        {1.0006 * 8.854e-12, 1.2566e-6, 0.0, 0.5f, 0.5f, 1.0f, "Air"},  // AIR
        {4.5 * 8.854e-12, 1.2566e-6, 1e-12, 0.3f, 0.8f, 1.0f, "Glass"}, // GLASS
        {8.854e-12, 1.2566e-6, 1e7, 0.8f, 0.8f, 0.8f, "Metal"},        // METAL
        {2.2 * 8.854e-12, 1.2566e-6, 1e-14,  1.0f, 0.8f, 0.6f,  "Teflon"},        // Low-loss microwave dielectric
        {10.0 * 8.854e-12, 1.2566e-6, 1e-4,  1.0f, 0.4f, 0.6f,  "Ceramic"},       // High-permittivity material
        {1.5 * 8.854e-12, 200 * 1.2566e-6, 1e-6, 0.2f, 0.9f, 0.2f, "Ferrite"},     // Magnetic material
        {2.5 * 8.854e-12, 1.2566e-6, 5.0e4,  1.0f, 1.0f, 0.4f,  "Salt Water"},    // High-conductivity, dispersive behavior
        {6.0 * 8.854e-12, 1.2566e-6, 1.0,    0.9f, 0.6f, 1.0f,  "Human Tissue"},
{-0.254e-12, -0.1566e-6, 1e4, 0.0f, 1.0f, 1.0f, "Metamaterial"}
    };

    

    // defaults
    state = (SimState){
        .mouseX = -1,
        .mouseY = -1,
        .mouseClicked = false,
        .shapeClicked = false,
        .amplitude = 10.0f,
        .boxSize = 50.0f,
        .dx = 0.5,
        .materials = material_list,
        .selected_material = 0,
        .num_materials = NUM_MATERIALS,
        .shape_type = 0
        };
    global_state = &state;

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowSize(SIZE_X, SIZE_Y);

    int main_window = glutCreateWindow("CUDA EM Sim");

    GLenum err = glewInit();
    if (err != GLEW_OK)
    {
        fprintf(stderr, "Error initializing GLEW: %s\n", glewGetErrorString(err));
        exit(1);
    }

    init_gpu(global_state);

    GLUI *glui = GLUI_Master.create_glui("Controls");

    glui->add_spinner("Simulation Step Speed (dx)", GLUI_SPINNER_FLOAT, &(global_state->dx), 1, control_cb)->set_float_limits(0.05, 1.0);
    glui->add_spinner("Pulse Amplitude", GLUI_SPINNER_FLOAT, &(global_state->amplitude), 1, control_cb)->set_float_limits(1.0, 100.0);
    glui->add_spinner("Pulse Spread", GLUI_SPINNER_FLOAT, &(global_state->spread), 1, control_cb)->set_float_limits(1.0, 100.0);

    glui->add_spinner("Box Size", GLUI_SPINNER_FLOAT, &(global_state->boxSize), 1, control_cb)->set_float_limits(5.0, 200.0);

    GLUI_Listbox *material_listbox = glui->add_listbox("Material Type", &(global_state->selected_material));
    for (int i = 0; i < global_state->num_materials; i++)
    {
        material_listbox->add_item(i, global_state->materials[i].name);
    }

    GLUI_Listbox *shape_type_listbox = glui->add_listbox("Shape Type", &(global_state->shape_type));
    shape_type_listbox->add_item(0, "Square");
    shape_type_listbox->add_item(1, "Circle");


    glui->set_main_gfx_window(main_window);

    create_pbo(global_state);

    glutDisplayFunc(display_wrapper);
    set_sim_state(global_state); // register state with input callbacks
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse_func);
    glutMotionFunc(motion_func);

    glutMainLoop();
    return 0;
}