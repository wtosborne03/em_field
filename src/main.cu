#include <GL/glew.h>
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <GL/glui.h>

#include "util/pbo.h"
#include "util/input.h"
#include "constants.h"
#include "sim.h"

static SimState *global_state = NULL;
static SimState state;

void display_wrapper()
{
    if (global_state)
    {
        display(global_state);
    }
}

int main(int argc, char **argv)
{

    // defaults
    state = (SimState){
        .mouseX = -1,
        .mouseY = -1,
        .mouseClicked = false,
        .amplitude = 10.0f,
        .boxSize = 50.0f};
    global_state = &state;

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowSize(SIZE_X, SIZE_Y);

    int main_window = glutCreateWindow("CUDA EM Sim");

    GLenum err = glewInit();
    if (err != GLEW_OK)
    {
        fprintf(stderr, "Error initializing GLEW: %s\n", glewGetErrorString(err));
        exit(1);
    }

    init_gpu(global_state);

    GLUI *glui = GLUI_Master.create_glui("Controls");

    glui->add_spinner("Amplitude", GLUI_SPINNER_FLOAT, &(global_state->amplitude), 1, control_cb)->set_float_limits(2.0, 50.0);
    glui->add_spinner("Box_Size", GLUI_SPINNER_FLOAT, &(global_state->boxSize), 1, control_cb)->set_float_limits(20.0, 200.0);
    glui->set_main_gfx_window(main_window);

    create_pbo(global_state);

    glutDisplayFunc(display_wrapper);
    set_sim_state(global_state); // register state with input callbacks
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse_func);

    glutMainLoop();
    return 0;
}