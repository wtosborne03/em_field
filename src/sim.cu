#include "hip/hip_runtime.h"
#include "sim.cuh"

const float dx = 0.01f;
const float mu0 = 4.0f * M_PI * 1e-7f; // permeability
const float eps0 = 8.854187817e-12f;   // permittivity
const float c = 1.0f / sqrt(mu0 * eps0);
const float dt = 0.99f * dx / (c * sqrtf(2.0f));
const float CeH = dt / (mu0 * dx);
const float CeE = dt / (eps0 * dx);

void init_gpu(SimState *state)
{
    size_t total_size = SIZE_X * SIZE_Y * sizeof(double);

    hipMalloc(&(state->d_Ez), total_size);
    hipMalloc(&(state->d_Hx), total_size);
    hipMalloc(&(state->d_Hy), total_size);
    // material grid alloc
    hipMalloc(&(state->d_epsilon), total_size);
    hipMalloc(&(state->d_mu), total_size);
    hipMalloc(&(state->d_sigma), total_size);


    hipMalloc(&(state->d_Ez_prev), total_size);  // same size as Ez

    //memset
    hipMemset(state->d_Ez, 0, total_size);
    hipMemset(state->d_Hx, 0, total_size);
    hipMemset(state->d_Hy, 0, total_size);



    hipMemset(state->d_epsilon, 0, total_size);
    hipMemset(state->d_mu, 0, total_size);
    hipMemset(state->d_sigma, 0, total_size);

    hipMemset(state->d_Ez_prev, 0, total_size);
    hipMalloc(&(state->d_field), sizeof(EM_field_d));

    EM_field_d h_field; // Host-side temporary struct
    h_field.Ez = state->d_Ez;
    h_field.Hx = state->d_Hx;
    h_field.Hy = state->d_Hy;
    h_field.epsilon = state->d_epsilon;
    h_field.mu = state->d_mu;
    h_field.sigma = state->d_sigma;

    // need to set initial material arrays on host before copying to device
    // needed for simulating air at start
    for (int i = 0; i < SIZE_X; i++) {
        for (int j = 0; j < SIZE_Y; j++) {
            h_field.epsilon[i * SIZE_X + j] = eps0;
            h_field.mu[i * SIZE_X + j] = mu0;
            h_field.sigma[i * SIZE_X + j] = 0.0;
        }
    }

    hipMemcpy(state->d_field, &h_field, sizeof(EM_field_d), hipMemcpyHostToDevice);
}

void display(SimState *state)
{
    size_t total_size = SIZE_X * SIZE_Y * sizeof(double);

    hipMemcpy(state->d_Ez_prev, state->d_Ez, total_size, hipMemcpyDeviceToDevice);

    dim3 block(16, 16), grid((SIZE_X + 15) / 16, (SIZE_Y + 15) / 16);

    updateH<<<grid, block>>>(state->d_field, SIZE_X, SIZE_Y, dt, dx);
    updateE<<<grid, block>>>(state->d_field, SIZE_X, SIZE_Y, dt, dx);
    apply_damping<<<(SIZE_X * SIZE_Y + 255) / 256, 256>>>(state->d_field, SIZE_X * SIZE_Y, 0.995f);

    mur_boundary<<<(SIZE_X * SIZE_Y + 255) / 256, 256>>>(state->d_Ez, SIZE_X, SIZE_Y, (c * dt / dx), state->d_Ez_prev);

    if (state->mouseClicked)
    {
        dim3 block(16, 16), grid((SIZE_X + 15) / 16, (SIZE_Y + 15) / 16);
        gaussian_pulse<<<grid, block>>>(state->d_field, SIZE_X, SIZE_Y, state->mouseX, SIZE_Y - state->mouseY, state->amplitude, 10.0f);
    }
    printf("MouseX: %i, MouseY: %i\n", state->mouseX, state->mouseY);

    float *d_pbo;
    size_t num_bytes;

    hipGraphicsMapResources(1, &(state->cuda_pbo_resource), 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_pbo, &num_bytes, state->cuda_pbo_resource);
    write_to_pbo<<<(SIZE_X * SIZE_Y + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(state->d_field,  d_pbo, SIZE_X * SIZE_Y);
    hipGraphicsUnmapResources(1, &(state->cuda_pbo_resource), 0);

    glClear(GL_COLOR_BUFFER_BIT);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, state->pbo);
    render_pbo();
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    glutSwapBuffers();
    glutPostRedisplay();
}

// deallocate before exiting
void cleanup(SimState *state)
{
    hipFree(state->d_Ez);
    hipFree(state->d_Hx);
    hipFree(state->d_Hy);
    hipFree(state->d_epsilon);
    hipFree(state->d_mu);
    hipFree(state->d_sigma);

    hipFree(state->d_field);
    hipFree(state->d_Ez_prev);
    hipGraphicsUnregisterResource(state->cuda_pbo_resource);
    glDeleteBuffers(1, &(state->pbo));
}